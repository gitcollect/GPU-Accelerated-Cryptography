/*
 * cudaUtil.cu
 *
 *  Created on: 2015-4-14
 *      Author: Yuqing Guan
 */

#include "util.h"

void gpuCheckError(hipError_t err, const char *file, int line)
{
	if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
